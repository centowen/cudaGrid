#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include "ndarray.h"

#include "DataIO.h"
#include "msio.h"
#include "Chunk.h"
#include "Coords.h"
#include <definitions.h>
#include "error.h"
using std::endl;
using std::cout;
using std::ios;
using std::fstream;


// Definitions of constants./*{{{*/
const int N_STOKES = 2;
const int THREADS = 128;
const int BLOCKS = 128;
const int chunk_size = 100000;
const int MAX_PHASE_CENTRES = 100;

// cuda constants
__constant__ float field_omega_u[MAX_PHASE_CENTRES];
__constant__ float field_omega_v[MAX_PHASE_CENTRES];
__constant__ float field_omega_w[MAX_PHASE_CENTRES];
/*}}}*/

typedef struct _DataContainer/*{{{*/
{
	float* u;
	float* v;
	float* w;
	float* freq;
	float* data_real;
	float* data_imag;
	float* data_weight;
	int* spw;
	int* field;
} DataContainer;/*}}}*/

typedef struct _DataGrid/*{{{*/
{
	float* vis_real;
	float* vis_imag;
	float* weight;
	float cell;
	size_t nx, ny;
	size_t nfields;
} DataGrid;/*}}}*/

// Function declarations./*{{{*/
__global__ void cudaGrid(DataContainer data, int chunk_size,
                         int nchan, DataGrid data_grid);
void grid(DataIO* dataio, DataGrid& data_grid, int mode, float x0, float y0);
void grid_to_numpy_containers(const char* vis,
                              Ndarray<double, 3> vis_real,
                              Ndarray<double, 3> vis_imag,
                              Ndarray<double, 3> weight,
                              Ndarray<double, 3> pb,
                              double cell, float x0, float y0,
							  int mode);

extern "C"{
const int grid_mode_uniform = 0;
const int grid_mode_natural = 1;
const int grid_mode_uniform_unweighted = 2;
void c_grid(const char* vis,
            numpyArray<double> c_vis_real, numpyArray<double> c_vis_imag,
            numpyArray<double> c_weight,
            numpyArray<double> c_pb,
            const double cell, const float x0, const float y0, 
			int mode = grid_mode_natural)
{
    Ndarray<double, 3> vis_real(c_vis_real);
    Ndarray<double, 3> vis_imag(c_vis_imag);
    Ndarray<double, 3> weight(c_weight);
    Ndarray<double, 3> pb(c_pb);

	grid_to_numpy_containers(vis, vis_real, vis_imag, weight, pb, cell, x0, y0, mode);
};
}

void allocate_cuda_data(DataContainer& data, DataGrid data_grid,
                        int nx, int ny, float cell, int nchan,
                        int nstokes, int chunk_size);

void setup(DataIO* dataio, 
           DataContainer& dev_data,
		   DataGrid& dev_data_grid, DataGrid& data_grid,
		   float x0, float y0);
void setup_freq(DataContainer& data, DataIO* dataio);
void setup_grid(DataGrid& data_grid, size_t nx, size_t ny, size_t nfields, float cell);
void setup_dev(DataGrid& data_grid, DataGrid& dev_data_grid,
		            DataIO* dataio, float x0, float y0);

int read_data_from_disk(DataIO* data, Chunk& chunk, clock_t& read_time);
void copy_data_to_cuda(DataContainer& data, Chunk& chunk);
void copy_grid_from_cuda(DataGrid& data_grid, DataGrid& dev_data_grid);
void normalize_grid(DataGrid& data_grid, const int mode);
void write_grid_to_disk(DataGrid& data_grid, const string& gridded_data_file,
                        const string& uvcoverage_file);

void reset_data_grid(DataGrid& data_grid);
void delete_grid(DataGrid& data_grid);
void cleanup(DataIO*& dataio, DataContainer& dev_data,
             DataGrid& dev_data_grid);
void cleanup_freq(DataContainer& dev_data);
void cleanup_grid(DataGrid& dev_data_grid);
void free_cuda_data(DataContainer& data);/*}}}*/

int main(int argc, char* argv[])/*{{{*/
{
	string vis;
	string gridded_data_file;
	string uvcoverage_file;
	string mode_string;
	DataGrid data_grid;
	int mode;

	if(argc >= 4)
	{
		vis = argv[1];
		gridded_data_file = argv[2];
		uvcoverage_file = argv[3];
	}
	else
	{
		cerr << "grid vis outdatafile outuvcov [gridmode]" << endl;
	}

	if( argc >= 5)
	{
		if(string(argv[4]) == "natural")
		{
			mode = grid_mode_natural;
			cout << "Using natural weighting." << endl;
		}
		else if(string(argv[4]) == "uniform")
		{
			mode = grid_mode_uniform;
			cout << "Using uniform weighting." << endl;
		}
		else if(string(argv[4]) == "uniform_unweighted")
			mode = grid_mode_uniform_unweighted;
		else
			mode = grid_mode_natural;
	}
	else
	{
		mode = grid_mode_natural;
	}

	DataIO* dataio = (DataIO*)new msio(vis.c_str(), "", true);
	setup_grid(data_grid, 64, 64, 1, 4.84813681109536e-06*0.2);
	grid(dataio, data_grid, mode, 0., 0.);
	delete_grid(data_grid);
}/*}}}*/

__global__ void cudaGrid(DataContainer data, int chunk_size, int nchan,/*{{{*/
                         DataGrid data_grid)
{
	int uvrow = threadIdx.x+blockIdx.x*blockDim.x;
	int grid_index, grid_index_inv;
	int u_index;
	int v_index;
	float weight_sum = 0.;
	float vis_real_sum = 0.;
	float vis_imag_sum = 0.;
	float phase_rot_phi;
	float phase_rot_real;
	float phase_rot_imag;

	while(uvrow < chunk_size && data.field[uvrow] < data_grid.nfields) // FIXME: Should check if field is being imaged.
	{
		float* freq = &data.freq[data.spw[uvrow]*nchan];
		for(int chanID = 0; chanID < nchan; chanID++)
		{
// 			u_index = int(data.u[uvrow]*freq[chanID]/c*data_grid.cell*data_grid.nx+data_grid.nx/2.);
// 			v_index = int(data.v[uvrow]*freq[chanID]/c*data_grid.cell*data_grid.ny+data_grid.ny/2.);
// 			u_index = int(-data.u[uvrow]*freq[chanID]/c*data_grid.cell/0.8859001628962232*data_grid.nx+data_grid.nx/2.+0.5);
			u_index = int(-data.u[uvrow]*freq[chanID]/c*data_grid.cell*data_grid.nx+data_grid.nx/2.+0.5);
			v_index = int(data.v[uvrow]*freq[chanID]/c*data_grid.cell*data_grid.ny+data_grid.ny/2.+0.5);

// 			grid_index = u_index*data_grid.ny + v_index;
// 			grid_index_inv = (data_grid.nx-u_index)*data_grid.ny + (data_grid.ny-v_index);
			grid_index = data.field[uvrow]*data_grid.nx*data_grid.ny + u_index*data_grid.ny + v_index;
			grid_index_inv = data.field[uvrow]*data_grid.nx*data_grid.ny + (data_grid.nx-u_index)*data_grid.ny + (data_grid.ny-v_index);

			phase_rot_phi = -freq[chanID]*(data.u[uvrow]*(field_omega_u[data.field[uvrow]])+
			                               data.v[uvrow]*(field_omega_v[data.field[uvrow]])+
			                               data.w[uvrow]*(field_omega_w[data.field[uvrow]]));
// 			phase_rot_phi = 0.;
			sincos(phase_rot_phi, &phase_rot_imag, &phase_rot_real);
// 			phase_rot_real = cosf(phase_rot_phi);
// 			phase_rot_imag = sinf(phase_rot_phi);

// 			if(chanID == 0 and uvrow == 0 and data.spw[uvrow] == 0)
// 			{
// 				atomicAdd(&data_grid.vis_real[29+30*data_grid.nx],
// 				          phase_rot_phi);
// 				atomicAdd(&data_grid.vis_real[30+30*data_grid.nx],
// 				          phase_rot_real);
// 				atomicAdd(&data_grid.vis_real[31+30*data_grid.nx],
// 				          phase_rot_imag);
// 				atomicAdd(&data_grid.vis_real[32+30*data_grid.nx],
// 				          1.);
// 				atomicAdd(&data_grid.vis_real[33+30*data_grid.nx],
// 				          data.u[uvrow]);
// 				atomicAdd(&data_grid.vis_real[34+30*data_grid.nx],
// 				          field_omega_u[data.field[uvrow]]);
// 				atomicAdd(&data_grid.vis_real[35+30*data_grid.nx],
// 				          freq[chanID]);
// 				atomicAdd(&data_grid.vis_real[36+30*data_grid.nx],
// 				          field_omega_u[0]);
// 			}

			for(int stokesID=0; stokesID < N_STOKES; stokesID++)
			{
				int weightindex = uvrow*N_STOKES + stokesID;
				int dataindex = nchan*(weightindex) + chanID;

				atomicAdd(&data_grid.vis_real[grid_index],
				          data.data_weight[weightindex]*(data.data_real[dataindex]*phase_rot_real-
						                                 data.data_imag[dataindex]*phase_rot_imag));
				atomicAdd(&data_grid.vis_imag[grid_index],
				          data.data_weight[weightindex]*(data.data_real[dataindex]*phase_rot_imag+
						                                 data.data_imag[dataindex]*phase_rot_real));
				atomicAdd(&data_grid.weight[grid_index], data.data_weight[weightindex]);

				atomicAdd(&data_grid.vis_real[grid_index_inv],
				          data.data_weight[weightindex]*(data.data_real[dataindex]*phase_rot_real-
						                                 data.data_imag[dataindex]*phase_rot_imag));
				atomicAdd(&data_grid.vis_imag[grid_index_inv],
				          -1.*data.data_weight[weightindex]*(data.data_real[dataindex]*phase_rot_imag+
						                                     data.data_imag[dataindex]*phase_rot_real));
				atomicAdd(&data_grid.weight[grid_index_inv], data.data_weight[weightindex]);

				weight_sum += data.data_weight[weightindex];
				vis_real_sum += data.data_real[dataindex]*data.data_weight[weightindex];
				vis_imag_sum += data.data_imag[dataindex]*data.data_weight[weightindex];

// 				atomicAdd(&data_grid.vis_real[data_grid.nx*data_grid.ny], data.data_real[dataindex]*data.data_weight[weightindex]);
// 				atomicAdd(&data_grid.vis_imag[data_grid.nx*data_grid.ny], data.data_imag[dataindex]*data.data_weight[weightindex]);
// 				atomicAdd(&data_grid.weight[data_grid.nx*data_grid.ny], data.data_weight[weightindex]);
			}
		}
		uvrow+=blockDim.x*gridDim.x; // Update the index of each thread by the number of threads launched simultaneosly (threads per block * number of blocks).
	}
}/*}}}*/
void grid(DataIO *dataio, DataGrid& data_grid,/*{{{*/
          const int mode, float x0, float y0)
{
	DataContainer data;
	DataGrid dev_data_grid;
	Chunk chunk(chunk_size);
	clock_t read_time = 0, gpu_time = 0, start, stop;

	setup(dataio, data, dev_data_grid, data_grid, x0, y0);

	while(read_data_from_disk( dataio,  chunk, read_time) > 0)
	{
		copy_data_to_cuda(data, chunk);
		start = clock();
		cudaGrid<<<BLOCKS,THREADS>>>(data, chunk.size(), dataio->nChan(), dev_data_grid);
		CudaCheckError();
		hipDeviceSynchronize();
		stop = clock();
		gpu_time += stop-start;
		cout << "*" << std::flush;
	}
	cout << endl;

	copy_grid_from_cuda(data_grid, dev_data_grid);
	cout << "Done copying data back!" << endl;

	normalize_grid(data_grid, mode);
	cout << "Done normalizing data." << endl;

	cout << "Time used to read data: " << (float)read_time / (float)CLOCKS_PER_SEC << endl;
	cout << "Time used in GPU: " << (float)gpu_time / (float)CLOCKS_PER_SEC << endl;


// 	cout.precision(10);
// 	cout.setf( std::ios::fixed, std:: ios::floatfield );

	cleanup(dataio, data, dev_data_grid);
}/*}}}*/

void allocate_cuda_data(DataContainer& data, const int nchan, const int nstokes, const int chunk_size)/*{{{*/
{
	CudaSafeCall(hipMalloc( (void**)&data.u, sizeof(float)*chunk_size));
	CudaSafeCall(hipMalloc( (void**)&data.v, sizeof(float)*chunk_size));
	CudaSafeCall(hipMalloc( (void**)&data.w, sizeof(float)*chunk_size));
	CudaSafeCall(hipMalloc( (void**)&data.data_real, sizeof(float)*chunk_size*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&data.data_imag, sizeof(float)*chunk_size*nchan*nstokes));
	CudaSafeCall(hipMalloc( (void**)&data.data_weight, sizeof(float)*chunk_size*nstokes));
	CudaSafeCall(hipMalloc( (void**)&data.spw, sizeof(int)*chunk_size));
	CudaSafeCall(hipMalloc( (void**)&data.field, sizeof(int)*chunk_size));
}/*}}}*/

void reset_data_grid(DataGrid& data_grid)/*{{{*/
{
	for(int i = 0; i < data_grid.nx*data_grid.ny+1; i++)
	{
		data_grid.vis_real[i] = 0.;
		data_grid.vis_imag[i] = 0.;
		data_grid.weight  [i] = 0.;
	}
}/*}}}*/

void setup_grid(DataGrid& data_grid, size_t nx, size_t ny,/*{{{*/
		        size_t nfields, float cell)
{
	data_grid.nx = nx;
	data_grid.ny = ny;
	data_grid.nfields = nfields;
	data_grid.cell = cell;
	data_grid.vis_real = new float[nx*ny*nfields+1];
	data_grid.vis_imag = new float[nx*ny*nfields+1];
	data_grid.weight = new float[nx*ny*nfields+1];
	reset_data_grid(data_grid);
}/*}}}*/

void setup(DataIO* dataio, /*{{{*/
           DataContainer& dev_data,
		   DataGrid& dev_data_grid, DataGrid& data_grid,
		   float x0, float y0)
{
	allocate_cuda_data(dev_data, dataio->nChan(), N_STOKES, chunk_size);
	setup_freq(dev_data, dataio);
	setup_dev(data_grid, dev_data_grid, dataio, x0, y0);
// 	setup_grid(data_grid, dev_data_grid, 512, 512, 4.84813681109536e-06*0.2*0.5);
// 	setup_grid(data_grid, dev_data_grid, 4096, 4096, 4.84813681109536e-06*0.2*0.5);
}/*}}}*/
void setup_freq(DataContainer& data, DataIO* dataio)/*{{{*/
{
	float* freq = new float[dataio->nChan()*dataio->nSpw()];
	CudaSafeCall(hipMalloc( (void**)&data.freq, sizeof(float)*dataio->nChan()*dataio->nSpw()));

	// Load frequencies into freq[].
	for(int chanID = 0; chanID < dataio->nChan(); chanID++)
	{
		for(int spwID = 0; spwID < dataio->nSpw(); spwID++)
		{
			freq[spwID*dataio->nChan()+chanID] = (float)dataio->getFreq(spwID)[chanID];
		}
	}
	CudaSafeCall(hipMemcpy(data.freq, freq,
	           sizeof(float)*dataio->nChan()*dataio->nSpw(),
	           hipMemcpyHostToDevice));
	delete[] freq;
}/*}}}*/
void setup_dev(DataGrid& data_grid, DataGrid& dev_data_grid, DataIO* dataio, float x0, float y0) /*{{{*/
{
	dev_data_grid.nx = data_grid.nx;
	dev_data_grid.ny = data_grid.ny;
	dev_data_grid.nfields = data_grid.nfields;
	dev_data_grid.cell = data_grid.cell;
	cout << "nfields: " << data_grid.nfields << endl;
	cout << "nfields: " << dev_data_grid.nfields << endl;
	cout << "grid size is " << 3*sizeof(float)*(dev_data_grid.nx*dev_data_grid.ny*dev_data_grid.nfields+1)/1024./1024 << " MiB." << endl;
	CudaSafeCall(hipMalloc( (void**)&dev_data_grid.vis_real,
	                         sizeof(float)*(dev_data_grid.nx*dev_data_grid.ny*dev_data_grid.nfields+1)));
	CudaSafeCall(hipMalloc( (void**)&dev_data_grid.vis_imag,
	                         sizeof(float)*(dev_data_grid.nx*dev_data_grid.ny*dev_data_grid.nfields+1)));
	CudaSafeCall(hipMalloc( (void**)&dev_data_grid.weight,
	                         sizeof(float)*(dev_data_grid.nx*dev_data_grid.ny*dev_data_grid.nfields+1)));
	CudaSafeCall(hipMemcpy(dev_data_grid.vis_real, data_grid.vis_real,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dev_data_grid.vis_imag, data_grid.vis_imag,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dev_data_grid.weight, data_grid.weight,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyHostToDevice));

	// Set up pointing information.
	int n_phase_centres = dataio->nPointings();
	if( n_phase_centres > MAX_PHASE_CENTRES)
	{
		std::cerr << "To many pointings for CUDA! Only taking " 
		          << n_phase_centres << " first." << endl;
		n_phase_centres = MAX_PHASE_CENTRES;
	}
	float *host_field_omega_u = new float[n_phase_centres];
	float *host_field_omega_v = new float[n_phase_centres];
	float *host_field_omega_w = new float[n_phase_centres];
	for(int i = 0; i < n_phase_centres; i++)
	{
		float dx = sin(x0 - dataio->xPhaseCentre(i)) * cos(y0);
		float dy = sin(y0)*cos(dataio->yPhaseCentre(i)) -
			       cos(y0)*sin(dataio->yPhaseCentre(i)) *
				   cos(x0-dataio->xPhaseCentre(i));
		dx = fmod(dx, (float)(2*M_PI));
		host_field_omega_u[i] = 2*M_PI*dx/c;
		host_field_omega_v[i] = 2*M_PI*dy/c;
		host_field_omega_w[i] = 2*M_PI*(sqrt(1-dx*dx-dy*dy)-1)/c;
		if(i == 0)
		{
			cout.precision(20);
			cout << "(x0, y0) = " << x0 << ", " << y0 << endl;
			cout << "phase centre: " << dataio->xPhaseCentre(i) << ", " << dataio->yPhaseCentre(i) << endl;
			cout << "(dx, dy) = " << dx*180*3600/M_PI << ", " << dy*180*3600/M_PI << endl;
			cout << "(omega_u, omega_v) = " << host_field_omega_u[0] << ", " << host_field_omega_v[0] << endl;
		}
	}

	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL( field_omega_u), host_field_omega_u,
	                                 sizeof(float)*n_phase_centres, 0,
	                                 hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL( field_omega_v), host_field_omega_v,
	                                 sizeof(float)*n_phase_centres, 0,
	                                 hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL( field_omega_w), host_field_omega_w,
	                                 sizeof(float)*n_phase_centres, 0,
	                                 hipMemcpyHostToDevice));

	delete[] host_field_omega_u;
	delete[] host_field_omega_v;
	delete[] host_field_omega_w;
}/*}}}*/

int read_data_from_disk(DataIO* data, Chunk& chunk, clock_t& read_time)/*{{{*/
{
	clock_t start, stop;
	int nrow;

	start = clock();
	nrow = data->readChunk(chunk);
	stop = clock();
	read_time += stop-start;
	return nrow;
}/*}}}*/
void copy_data_to_cuda(DataContainer& data, Chunk& chunk)/*{{{*/
{
	int chunk_size = chunk.size();
	float* u = new float[chunk_size];
	float* v = new float[chunk_size];
	float* w = new float[chunk_size];
	int *spw = new int[chunk_size];
	int *field = new int[chunk_size];
	for(int uvrow = 0; uvrow < chunk_size; uvrow++)
	{
		u[uvrow] = chunk.inVis[uvrow].u;
		v[uvrow] = chunk.inVis[uvrow].v;
		w[uvrow] = chunk.inVis[uvrow].w;
		spw[uvrow] = chunk.inVis[uvrow].spw;
		field[uvrow] = chunk.inVis[uvrow].fieldID;
	}

// 	cout << "u,v: " << u[0] << ", " << v[0] << endl;
// 	cout << "freq: " << chunk.inVis[0].freq[0] << endl;

	CudaSafeCall(hipMemcpy(data.u, u, sizeof(float)*chunk.size(), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.v, v, sizeof(float)*chunk.size(), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.w, w, sizeof(float)*chunk.size(), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.spw, spw, sizeof(float)*chunk.size(), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.field, field, sizeof(float)*chunk.size(), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.data_real,   chunk.data_real_in,
				sizeof(float)*chunk.size()*chunk.nChan()*N_STOKES,
				hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.data_imag,   chunk.data_imag_in,
				sizeof(float)*chunk.size()*chunk.nChan()*N_STOKES,
				hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(data.data_weight, chunk.weight_in,
				sizeof(float)*chunk.size()*N_STOKES,
				hipMemcpyHostToDevice));
	delete[] u;
	delete[] v;
	delete[] w;
	delete[] spw;
	delete[] field;
}/*}}}*/
void copy_grid_from_cuda(DataGrid& data_grid, DataGrid& dev_data_grid)/*{{{*/
{
	CudaSafeCall(hipMemcpy(data_grid.vis_real, dev_data_grid.vis_real,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(data_grid.vis_imag, dev_data_grid.vis_imag,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(data_grid.weight, dev_data_grid.weight,
	                        sizeof(float)*(data_grid.nx*data_grid.ny*data_grid.nfields+1),
	                        hipMemcpyDeviceToHost));
}/*}}}*/
void normalize_grid(DataGrid& data_grid, const int mode)/*{{{*/
{
	if( mode == grid_mode_natural)
	{
		float sum_of_weights;
		for(int field = 0; field < data_grid.nfields; field++)
		{
			sum_of_weights = 0;
			for(int i = 0; i < data_grid.nx*data_grid.ny; i++)
					sum_of_weights += data_grid.weight[i+field*data_grid.nx*data_grid.ny];

			for(int i = 0; i < data_grid.nx*data_grid.ny; i++)
			{
				data_grid.vis_real[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
				data_grid.vis_imag[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
				data_grid.weight[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
			}
		}
	}
	else if( mode == grid_mode_uniform )
	{
		float sum_of_weights = 0.;

		for(int field = 0; field < data_grid.nfields; field++)
		{
			sum_of_weights = 0.;
			for(int i = 0; i < data_grid.nx*data_grid.ny; i++)
			{
				if(data_grid.weight[i+field*data_grid.nx*data_grid.ny] > 0)
				{
					data_grid.vis_real[i+field*data_grid.nx*data_grid.ny] /= data_grid.weight[i+field*data_grid.nx*data_grid.ny];
					data_grid.vis_imag[i+field*data_grid.nx*data_grid.ny] /= data_grid.weight[i+field*data_grid.nx*data_grid.ny];
					data_grid.weight[i+field*data_grid.nx*data_grid.ny] = 1.;
					sum_of_weights += 1.;
				}
			}

			for(int i = 0; i < data_grid.nx*data_grid.ny; i++)
			{
				data_grid.vis_real[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
				data_grid.vis_imag[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
				data_grid.weight[i+field*data_grid.nx*data_grid.ny] /= sum_of_weights;
			}
		}
	}
}/*}}}*/

void cleanup(DataIO*& dataio, DataContainer& dev_data,/*{{{*/
             DataGrid& dev_data_grid)
{
	cleanup_freq(dev_data);
	cleanup_grid(dev_data_grid);
	free_cuda_data(dev_data);
	delete dataio;
}/*}}}*/
void cleanup_freq(DataContainer& dev_data)/*{{{*/
{
	CudaSafeCall(hipFree(dev_data.freq));
}/*}}}*/
void delete_grid(DataGrid& data_grid)/*{{{*/
{
	delete[] data_grid.vis_real;
	delete[] data_grid.vis_imag;
	delete[] data_grid.weight;
	data_grid.vis_real = NULL;
	data_grid.vis_imag = NULL;
	data_grid.weight = NULL;
}/*}}}*/
void cleanup_grid(DataGrid& dev_data_grid)/*{{{*/
{
	CudaSafeCall(hipFree(dev_data_grid.vis_real));
	CudaSafeCall(hipFree(dev_data_grid.vis_imag));
	CudaSafeCall(hipFree(dev_data_grid.weight));
	dev_data_grid.vis_real = NULL;
	dev_data_grid.vis_imag = NULL;
	dev_data_grid.weight = NULL;
}/*}}}*/
void free_cuda_data(DataContainer& data)/*{{{*/
{
	CudaSafeCall(hipFree( data.u));
	CudaSafeCall(hipFree( data.v));
	CudaSafeCall(hipFree( data.w));
	CudaSafeCall(hipFree( data.data_real));
	CudaSafeCall(hipFree( data.data_imag));
	CudaSafeCall(hipFree( data.data_weight));
	CudaSafeCall(hipFree( data.spw));
}/*}}}*/
void grid_to_numpy_containers(const char* vis, /*{{{*/
                              Ndarray<double, 3> vis_real,
                              Ndarray<double, 3> vis_imag,
                              Ndarray<double, 3> weight,
                              Ndarray<double, 3> pb,
                              double cell, float x0, float y0,
							  int mode)
{
	DataGrid data_grid;
	DataIO* dataio = (DataIO*)new msio(vis, "", true);

	setup_grid(data_grid, vis_real.getShape(1), vis_real.getShape(2), vis_real.getShape(0),
		       float(cell));

// 	grid((string)vis, data_grid, mode);
	grid(dataio, data_grid, mode, x0, y0);

// 	cout.precision(30);
// 	cout << "V(29,30): " << data_grid.vis_real[29+30*64] << endl;
// 	cout << "V(30,30): " << data_grid.vis_real[30+30*64] << endl;
// 	cout << "V(31,30): " << data_grid.vis_real[31+30*64] << endl;
// 	cout << "V(32,30): " << data_grid.vis_real[32+30*64] << endl;
// 	cout << "V(33,30): " << data_grid.vis_real[33+30*64] << endl;
// 	cout << "V(34,30): " << data_grid.vis_real[34+30*64] << endl;
// 	cout << "V(35,30): " << data_grid.vis_real[35+30*64] << endl;
// 	cout << "V(36,30): " << data_grid.vis_real[36+30*64] << endl;

	int len = vis_real.getShape(0)*vis_real.getShape(1)*vis_real.getShape(2);

    std::copy(data_grid.vis_real, &data_grid.vis_real[len], vis_real.begin());
    std::copy(data_grid.vis_imag, &data_grid.vis_imag[len], vis_imag.begin());
    std::copy(data_grid.weight, &data_grid.weight[len], weight.begin());
	delete_grid(data_grid);
}/*}}}*/
void write_grid_to_disk(DataGrid& data_grid, const string& gridded_data_file,/*{{{*/
		const string& uvcoverage_file)
{
	fstream datafile(gridded_data_file.c_str(), ios::out);
	fstream uvcovfile(uvcoverage_file.c_str(), ios::out);

	for(int uindex = 0; uindex < data_grid.nx; uindex++)
	{
		for(int vindex = 0; vindex < data_grid.ny; vindex++)
		{
			datafile << data_grid.vis_real[uindex+data_grid.nx*vindex];
			if(data_grid.vis_imag[uindex+data_grid.nx*vindex] >= 0.)
				datafile << "+";
			datafile << data_grid.vis_imag[uindex+data_grid.nx*vindex] << "j ";
		}
		datafile << "\n";
	}
	for(int uindex = 0; uindex < data_grid.nx; uindex++)
	{
		for(int vindex = 0; vindex < data_grid.ny; vindex++)
		{
			uvcovfile << data_grid.weight[uindex+data_grid.nx*vindex] << " ";
		}
		uvcovfile << "\n";
	}
	datafile.close();
	uvcovfile.close();
	cout << "Done writing data." << endl;
}/*}}}*/
